
#include <hip/hip_runtime.h>
void __global__ calcU_GPU(float3* r, float* u){

	//float3* r = (float3*) r_data;
	//float* u = (float*) u_data;

	//if(threadIdx.x != blockIdx.x){

	if(blockIdx.x != threadIdx.x){

		float3 r_i = r[blockIdx.x];
		float3 r_j = r[threadIdx.x];

		int pointer = blockDim.x * blockIdx.x * 9 + (threadIdx.x * 9);
		//cuPrintf("%d\n",pointer);
		//	cuPrintf("BLOCK: %d pointer= %d\n",blockDim.x,pointer);
		for(int i=0;i<blockIdx.x;i++){

			float r1 = sqrt(pow(r_i.x-r_j.x,2) + pow(r_i.y-r_j.y,2) + pow(r_i.z-r_j.z,2));

			// x
			u[pointer]     = (3 * (r_i.x - r_j.x)/r1 * (r_i.x - r_j.x)/r1 - 1) / r1*r1*r1;
			u[pointer + 1] = (3 * (r_i.x - r_j.x)/r1 * (r_i.y - r_j.y)/r1    ) / r1*r1*r1;
			u[pointer + 2] = (3 * (r_i.x - r_j.x)/r1 * (r_i.z - r_j.z)/r1    ) / r1*r1*r1;

			// y
			u[pointer + 3] = (3 * (r_i.y - r_j.y)/r1 * (r_i.x - r_j.x)/r1    ) / r1*r1*r1;
			u[pointer + 4] = (3 * (r_i.y - r_j.y)/r1 * (r_i.y - r_j.y)/r1 - 1) / r1*r1*r1;
			u[pointer + 5] = (3 * (r_i.y - r_j.y)/r1 * (r_i.z - r_j.z)/r1    ) / r1*r1*r1;

			// z
			u[pointer + 6] = (3 * (r_i.z - r_j.z)/r1 * (r_i.x - r_j.x)/r1    ) / r1*r1*r1;
			u[pointer + 7] = (3 * (r_i.z - r_j.z)/r1 * (r_i.y - r_j.y)/r1    ) / r1*r1*r1;
			u[pointer + 8] = (3 * (r_i.z - r_j.z)/r1 * (r_i.z - r_j.z)/r1 - 1) / r1*r1*r1;


			}
		}
	}
