#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cuPrintf.cu>
#include <float.h>
#include "kernels.cu"
#include "naive.h"
#include <string.h>




//#define eps_f FLT_EPSILON




int main(int argc, char** argv){

	size_t buff=9999999;
	cudaPrintfInit(buff);

	if(argc < 2){
		puts("neoNbody: Usage: neoNbody numPoints");
		exit(0);
		}


	int L_SIZE = atoi(argv[1]);
	int N_POINTS = L_SIZE * L_SIZE;

	printf("%d %d\n",L_SIZE,N_POINTS);

	hipError_t status = hipSuccess;


// Alloc CPU memory

	float3* r_CPU;
	r_CPU = (float3*) malloc(sizeof(float3) * N_POINTS * N_POINTS);

	float* u_CPU;
	u_CPU = (float*) malloc(sizeof(float) * 3 * 3 * N_POINTS * N_POINTS);

	

	fillR_CPU(r_CPU,L_SIZE);

	
// Alloc GPU Memory


	float3 *r_GPU;
	status=hipMalloc((void**) &r_GPU, sizeof(float3) * N_POINTS);
	if (status != hipSuccess) 
		fprintf(stderr,"hipMalloc: allocating r: %s\n", status);

	status=hipMemcpy(r_GPU,r_CPU,sizeof(float3) * N_POINTS,hipMemcpyHostToDevice);
	if (status != hipSuccess) fprintf(stderr, "hipMemcpy: r: copy from Host to Device\n",status);


	float *u_GPU;
	status=hipMalloc((void**) &u_GPU, sizeof(float) * N_POINTS * N_POINTS * 3 * 3);
	if (status != hipSuccess) 
		fprintf(stderr,"hipMalloc: allocating u: %s\n", status);


// Calculate U

	dim3 blcsGrid(L_SIZE,1);
	dim3 thrdsBlck(L_SIZE,1);

	calcU_GPU<<<blcsGrid,thrdsBlck>>>(r_GPU,u_GPU);


	status=hipMemcpy(u_CPU,u_GPU,sizeof(float) * N_POINTS * N_POINTS * 3 * 3, hipMemcpyDeviceToHost);
	if (status != hipSuccess) 
	fprintf(stderr,"hipMemcpy: copy from Device to Host u: %s\n", status);
/*
	for(int i=0;i<N_POINTS*9;i++){
		printf("%d\n",i/9);
		printf("%f %f %f\n",u_CPU[i++],u_CPU[i++],u_CPU[i++]);
		printf("%f %f %f\n",u_CPU[i++],u_CPU[i++],u_CPU[i++]);
		printf("%f %f %f\n",u_CPU[i++],u_CPU[i++],u_CPU[i]);
		puts("");
		}
*/

	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	}